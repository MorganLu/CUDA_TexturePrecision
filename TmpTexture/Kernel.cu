#include "hip/hip_runtime.h"
#include "Kernel.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>


static texture<float4, 2, hipReadModeElementType> texElnt;
static hipArray	*texElntArray = NULL;
float  *pTmpElntArray = NULL;
float  *hArrResult    = NULL;
float  *dArrResult    = NULL;

extern "C" int InitTex( float *pData, int width, int height, int channel ) {
	int cn = UNIT*3;
	int fn = width*height*4;
	hArrResult     = new float  [cn];
	checkCudaErrors( hipMalloc( (void**)&dArrResult, (cn)*sizeof(float) ) );

	pTmpElntArray  = new float [fn];
	memset(pTmpElntArray, 0, fn*sizeof(float));

	float *ptrData = pData;
	float *ptrElnt = pTmpElntArray;

	for (int i=0;i<width*height;i++) {
		for (int c=0;c<channel;c++, ptrData++) {
			(*(ptrElnt+c)) = (float) (*(ptrData));
		}
		ptrElnt += 4;
	}


	hipChannelFormatDesc channelDesc; 
	channelDesc = hipCreateChannelDesc<float4>();
	checkCudaErrors( hipMallocArray(&texElntArray, &channelDesc, width, height) );
	checkCudaErrors( hipMemcpy2DToArray(texElntArray, 0, 0, pTmpElntArray, width*sizeof(float4), width*sizeof(float4), height, hipMemcpyHostToDevice) );

	texElnt.addressMode[0] = hipAddressModeClamp;
	texElnt.addressMode[1] = hipAddressModeClamp;
	texElnt.filterMode = hipFilterModeLinear;
	texElnt.normalized = false;

	checkCudaErrors( hipUnbindTexture(texElnt) );
	checkCudaErrors( hipBindTextureToArray(texElnt, texElntArray, channelDesc) );
	return 0;
}

static __global__ void kernel_texElnt(float* pdata, int w, int h, int c, float stride) {
	const int gx = blockIdx.x*blockDim.x + threadIdx.x;
	const int gy = blockIdx.y*blockDim.y + threadIdx.y;
	const int gw = gridDim.x * blockDim.x;
	const int gid = gy*gw + gx;
	float2 pnt;
	pnt.x = (gx)*(stride);
	pnt.y = 0.0625f;

	float4 result = tex2D( texElnt, pnt.x + 0.5, pnt.y + 0.5f);
	pdata[gid*3 + 0] = pnt.x;
	pdata[gid*3 + 1] = pnt.y;
	pdata[gid*3 + 2] = result.x;

}

extern "C" int RunKernel( int w, int h, int c, float nBase) {
	float stride = 1.0f / nBase;
	kernel_texElnt<<< 1, UNIT >>> (dArrResult, w, h, c, stride);
	checkCudaErrors( hipMemcpy(hArrResult, dArrResult, UNIT*3*sizeof(float), hipMemcpyDeviceToHost) );
	return 0;
}

extern "C" int UnInitTex() {
	delete hArrResult;
	delete pTmpElntArray;
	checkCudaErrors( hipFree(dArrResult) );
	checkCudaErrors( hipFreeArray(texElntArray) );
	return 0;
}